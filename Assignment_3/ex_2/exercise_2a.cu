
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <limits>
#include <sys/time.h>

// #define PINNED_MEMORY

#define NUM_PARTICLES 100000
#define NUM_ITERATIONS 1000
#define BLOCK_SIZE 256

#define MICROSECONDS(start, end) ((end.tv_sec - start.tv_sec) * 1000000LL + end.tv_usec - start.tv_usec)
#define MILLISECONDS(start, end) MICROSECONDS(start, end) / 1000.0
#define SECONDS(start, end) MILLISECONDS(start, end) / 1000.0

typedef struct
{
    float3 position;
    float3 velocity;
} Particle;

void cpu_timestep(Particle *particles, const float dt)
{
    for (unsigned int i = 0; i < NUM_PARTICLES; i++)
    {
        particles[i].position.x += particles[i].velocity.x * dt;
        particles[i].position.y += particles[i].velocity.y * dt;
        particles[i].position.z += particles[i].velocity.z * dt;
    }
}

__global__ void gpu_timestep(Particle *particles, const float dt)
{
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < NUM_PARTICLES)
    {
        particles[i].position.x += particles[i].velocity.x * dt;
        particles[i].position.y += particles[i].velocity.y * dt;
        particles[i].position.z += particles[i].velocity.z * dt;
    }
}

int main(int argc, char **argv)
{
    struct timeval start, end;
    const float dt = 1.0;

    // Initialize CPU data.
#ifdef PINNED_MEMORY
    Particle *cpu_particles;
    cudaMallocHost(&cpu_particles, NUM_PARTICLES * sizeof(Particle));
#else
    Particle *cpu_particles = (Particle *)malloc(NUM_PARTICLES * sizeof(Particle));
#endif

    for (unsigned int i = 0; i < NUM_PARTICLES; i++)
    {
        cpu_particles[i].position.x = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        cpu_particles[i].position.y = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        cpu_particles[i].position.z = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        cpu_particles[i].velocity.x = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        cpu_particles[i].velocity.y = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        cpu_particles[i].velocity.z = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    }

    // Initialize data.
    Particle *gpu_particles;
    hipMalloc(&gpu_particles, NUM_PARTICLES * sizeof(Particle));

    // Run simulation.
    printf("Running simulation... ");
    gettimeofday(&start, NULL);
    for (unsigned int i = 0; i < NUM_ITERATIONS; i++)
    {
        hipMemcpy(gpu_particles, cpu_particles, NUM_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);
        gpu_timestep<<<(NUM_PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(gpu_particles, dt);
        hipDeviceSynchronize();
        hipMemcpy(cpu_particles, gpu_particles, NUM_PARTICLES * sizeof(Particle), hipMemcpyDeviceToHost); // Copy anywhere.
    }
    gettimeofday(&end, NULL);
    printf("Done! Took %lfs.\n", SECONDS(start, end));

    // Free resources.
#ifdef PINNED_MEMORY
    cudaFreeHost(cpu_particles);
#else
    free(cpu_particles);
#endif
    hipFree(gpu_particles);

    return 0;
}